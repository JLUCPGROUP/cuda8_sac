#include "hip/hip_runtime.h"
#pragma once

#include <iostream>
#include <string>
#include <time.h>
#include <windows.h>
#include "model/XBuilder.h"
#include "model/HBuilder.h"
#include "cuda_sac.cuh"
#include "model/Timer.h"

#undef DOMDocument

using namespace std;
using namespace cudacp;
const string X_PATH = "BMPath.xml";

int main() {

	DWORD t1, t2;
	clock_t  begin, end;
	time_t start, end2;
	XBuilder path_builder(X_PATH, XRT_BM_PATH);
	string bm_path = path_builder.GetBMFile();
	cout << bm_path << endl;
	XBuilder builder(bm_path, XRT_BM);
	XModel* xmodel = new XModel();
	builder.GenerateModelFromXml(xmodel);
	HModel* hmodel = new HModel();
	HBuilder hbuilder(xmodel, hmodel);
	//hbuilder.ShowHModel();
	std::cout << "----------------modeling--------------------" << std::endl;
	float build_time = BuidBitModel32bit(hmodel);
	printf("Build time = %f\n", build_time);
	float exe_time = SACGPU();
	printf("Execution time = %f\n", exe_time);
	float back_time = CopyBitSubDom();
	printf("Copy back time = %f\n", back_time);
	std::cout << "----------------modeling--------------------" << std::endl;
	Solver s(hmodel);
	//Timer t;
	//begin = clock();
	std::cout << "----------------solving---------------------" << std::endl;
	t1 = GetTickCount();
	float sol_time = s.Solve(SM_DFS, SN_ONE);
	t2 = GetTickCount();
	//end = clock();
	std::cout << "----------------solving---------------------" << std::endl;
	//int64_t dur = t.elapsed();
	printf("Solve time = %f\n", (t2 - t1)*1.0);
	printf("nodes = %d\n", s.num_nodes);
	printf("|solutions| = %d\n", s.num_solutions);
	DelGPUModel();
	delete hmodel;
	delete xmodel;
	printf("---end---\n");
	return 0;
}

